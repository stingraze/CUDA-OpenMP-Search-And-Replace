#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hybrid_search_replace.cuh"
#include <string>
#include <vector>

// CUDA kernel for search and replace
__global__ void search_and_replace(char *text, size_t text_length, const char *search, const char *replace, size_t search_length, size_t replace_length) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx + search_length > text_length) return;

    // Check for a match
    bool match = true;
    for (size_t i = 0; i < search_length; ++i) {
        if (text[idx + i] != search[i]) {
            match = false;
            break;
        }
    }

    // Perform replacement if there's a match
    if (match) {
        for (size_t i = 0; i < replace_length; ++i) {
            text[idx + i] = replace[i];
        }
        // Null-terminate if replacement is shorter
        for (size_t i = replace_length; i < search_length; ++i) {
            text[idx + i] = '\0';
        }
    }
}

void process_chunk_with_cuda(std::vector<char> &chunk, const std::string &search, const std::string &replace) {
    size_t chunk_size = chunk.size();

    // Allocate memory on GPU
    char *d_text;
    hipMalloc(&d_text, chunk_size);
    hipMemcpy(d_text, chunk.data(), chunk_size, hipMemcpyHostToDevice);

    char *d_search, *d_replace;
    size_t search_length = search.size();
    size_t replace_length = replace.size();
    hipMalloc(&d_search, search_length);
    hipMalloc(&d_replace, replace_length);
    hipMemcpy(d_search, search.c_str(), search_length, hipMemcpyHostToDevice);
    hipMemcpy(d_replace, replace.c_str(), replace_length, hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threads_per_block = 256;
    int blocks_per_grid = (chunk_size + threads_per_block - 1) / threads_per_block;
    search_and_replace<<<blocks_per_grid, threads_per_block>>>(d_text, chunk_size, d_search, d_replace, search_length, replace_length);

    // Copy the result back to CPU
    hipMemcpy(chunk.data(), d_text, chunk_size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_text);
    hipFree(d_search);
    hipFree(d_replace);
}

